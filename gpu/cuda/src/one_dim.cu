
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

__global__ void kernel(int *array)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;

  array[index] = index;
}

int main(void)
{
  int num_elements = 256;

  int num_bytes = num_elements * sizeof(int);

  // pointers to host & device arrays
  int *device_array = 0;
  int *host_array = 0;

  // malloc a host array
  host_array = (int*)malloc(num_bytes);

  // cudaMalloc a device array
  hipMalloc((void**)&device_array, num_bytes);

  int block_size = 16;
  int grid_size = num_elements / block_size;

  kernel<<<grid_size,block_size>>>(device_array);

  // download and inspect the result on the host:
  hipMemcpy(host_array, device_array, num_bytes, hipMemcpyDeviceToHost);

  // print out the result element by element
  for(int i=0; i < num_elements; ++i)
  {
    printf("%3d ", host_array[i]);
    if((i+1) % block_size == 0)
        printf("\n");
  }
 
  // deallocate memory
  free(host_array);
  hipFree(device_array);
}

