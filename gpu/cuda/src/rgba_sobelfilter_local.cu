#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

#define BPP 4

#define BLOCKSIZE 16
#define TILESIZE 18


__shared__ short int Gx[3][3];
__shared__ short int Gy[3][3];

__shared__ uchar4 localBlock[TILESIZE*TILESIZE]; // BLOCKSIZE=16; TILESIZE=18

__device__ int sobel(unsigned char k00, unsigned char k01, unsigned char k02,
                     unsigned char k10, unsigned char k11, unsigned char k12,
                     unsigned char k20, unsigned char k21, unsigned char k22)
{
    Gx[0][0]=-1; Gx[0][1]=0; Gx[0][2]=1;
    Gx[1][0]=-2; Gx[1][1]=0; Gx[1][2]=2;
    Gx[2][0]=-1; Gx[2][1]=0; Gx[2][2]=1;
    Gy[0][0]=1; Gy[0][1]=2; Gy[0][2]=1;
    Gy[1][0]=0; Gy[1][1]=0; Gy[1][2]=0;
    Gy[2][0]=-1; Gy[2][1]=-2; Gy[2][2]=-1;
    int pixelX = k00*Gx[0][0] + k01*Gx[0][1] + k02*Gx[0][2] +
                 k10*Gx[1][0] + k11*Gx[1][1] + k12*Gx[1][2] +
                 k20*Gx[2][0] + k21*Gx[2][1] + k22*Gx[2][2];
    int pixelY = k00*Gy[0][0] + k01*Gy[0][1] + k02*Gy[0][2] +
                 k10*Gy[1][0] + k11*Gy[1][1] + k12*Gy[1][2] +
                 k20*Gy[2][0] + k21*Gy[2][1] + k22*Gy[2][2];
    int pixel = abs(pixelX)+abs(pixelY);
    pixel=(pixel<0)?0:pixel;
    pixel=(pixel>255)?255:pixel;

    return pixel;
}

__device__ int getindexForPixelAt(int x, int y, int width, int height)
{
    int val = x + y*width;
    const int max = width*height-1;
    if (val < 0)
    {
        return 0;
    }
    if (val > max)
    {
        return max;
    }
    return val;
}

__device__ uchar4 pixelAt(uchar4 *localBlock, int x, int y, int width,int height)
{
    int start = getindexForPixelAt(x, y, width, height);

    return localBlock[start];
}

__global__ void local_filter(unsigned char* input,uchar4* output,int width, int height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    int tx = threadIdx.x+1; int ty = threadIdx.y+1;
    int w = blockDim.y+2; int h = blockDim.x+2;

    uchar4 k00,k01,k02,k10,k11,k12,k20,k21,k22;
    uchar4 pixel = {0,0,0,255};
    //pixel.x = 0, pixel.y = 0, pixel.z = 0, pixel.w = 255;

    if(tx>=1 && ty>=1 && tx<=TILESIZE-2 && ty<=TILESIZE-2){
        localBlock[tx+ty*w].x = input[BPP*(x+y*width)];
        localBlock[tx+ty*w].y = input[BPP*(x+y*width)+1];
        localBlock[tx+ty*w].z = input[BPP*(x+y*width)+2];
        localBlock[tx+ty*w].w = input[BPP*(x+y*width)+3];
    }
    if(tx-1==0 && ty-1==0){
        localBlock[tx-1+(ty-1)*w].x = input[BPP*(x-1+(y-1)*width)];
        localBlock[tx-1+(ty-1)*w].y = input[BPP*(x-1+(y-1)*width)+1];
        localBlock[tx-1+(ty-1)*w].z = input[BPP*(x-1+(y-1)*width)+2];
        localBlock[tx-1+(ty-1)*w].w = input[BPP*(x-1+(y-1)*width)+3];
    }
    if(tx-1==0 && ty>=1 && ty<=TILESIZE-2){
        localBlock[tx-1+ty*w].x = input[BPP*(x-1+y*width)];
        localBlock[tx-1+ty*w].y = input[BPP*(x-1+y*width)+1];
        localBlock[tx-1+ty*w].z = input[BPP*(x-1+y*width)+2];
        localBlock[tx-1+ty*w].w = input[BPP*(x-1+y*width)+3];
    }
    if(tx-1==0 && ty+1==TILESIZE-1){
        localBlock[tx-1+(ty+1)*w].x = input[BPP*(x-1+(y+1)*width)];
        localBlock[tx-1+(ty+1)*w].y = input[BPP*(x-1+(y+1)*width)+1];
        localBlock[tx-1+(ty+1)*w].z = input[BPP*(x-1+(y+1)*width)+2];
        localBlock[tx-1+(ty+1)*w].w = input[BPP*(x-1+(y+1)*width)+3];
    }
    if(ty-1==0 && tx+1==TILESIZE-1){
        localBlock[tx+1+(ty-1)*w].x = input[BPP*(x+1+(y-1)*width)];
        localBlock[tx+1+(ty-1)*w].y = input[BPP*(x+1+(y-1)*width)+1];
        localBlock[tx+1+(ty-1)*w].z = input[BPP*(x+1+(y-1)*width)+2];
        localBlock[tx+1+(ty-1)*w].w = input[BPP*(x+1+(y-1)*width)+3];
    }
    if(ty>=1 && ty<=TILESIZE-2 && tx+1==TILESIZE-1){
        localBlock[tx+1+ty*w].x = input[BPP*(x+1+y*width)];
        localBlock[tx+1+ty*w].y = input[BPP*(x+1+y*width)+1];
        localBlock[tx+1+ty*w].z = input[BPP*(x+1+y*width)+2];
        localBlock[tx+1+ty*w].w = input[BPP*(x+1+y*width)+3];
    }
    if(ty+1==TILESIZE-1 && tx+1==TILESIZE-1){
        localBlock[tx+1+(ty+1)*w].x = input[BPP*(x+1+(y+1)*width)];
        localBlock[tx+1+(ty+1)*w].y = input[BPP*(x+1+(y+1)*width)+1];
        localBlock[tx+1+(ty+1)*w].z = input[BPP*(x+1+(y+1)*width)+2];
        localBlock[tx+1+(ty+1)*w].w = input[BPP*(x+1+(y+1)*width)+3];
    }
    if(ty-1==0 && tx>=1 && tx<=TILESIZE-2){
        localBlock[tx+(ty-1)*w].x = input[BPP*(x+(y-1)*width)];
        localBlock[tx+(ty-1)*w].y = input[BPP*(x+(y-1)*width)+1];
        localBlock[tx+(ty-1)*w].z = input[BPP*(x+(y-1)*width)+2];
        localBlock[tx+(ty-1)*w].w = input[BPP*(x+(y-1)*width)+3];
    }
    if(ty+1==TILESIZE-1 && tx>=1 && tx<=TILESIZE-2){
        localBlock[tx+(ty+1)*w].x = input[BPP*(x+(y+1)*width)];
        localBlock[tx+(ty+1)*w].y = input[BPP*(x+(y+1)*width)+1];
        localBlock[tx+(ty+1)*w].z = input[BPP*(x+(y+1)*width)+2];
        localBlock[tx+(ty+1)*w].w = input[BPP*(x+(y+1)*width)+3];
    }   
    __syncthreads();

    if(tx>=1 && tx<TILESIZE-1 && ty>=1 && ty<TILESIZE-1){
        k00 = pixelAt(localBlock, tx-1, ty-1, w, h);
        k01 = pixelAt(localBlock, tx-1, ty, w, h);
        k02 = pixelAt(localBlock, tx-1, ty+1, w, h);

        k10 = pixelAt(localBlock, tx, ty-1, w, h);
        k11 = pixelAt(localBlock, tx, ty, w, h);
        k12 = pixelAt(localBlock, tx, ty+1, w, h);

        k20 = pixelAt(localBlock, tx+1, ty-1, w, h);
        k21 = pixelAt(localBlock, tx+1, ty, w, h);
        k22 = pixelAt(localBlock, tx+1, ty+1, w, h);

        int gradient_r = sobel(k00.x, k01.x, k02.x, k10.x, k11.x, k12.x, k20.x, k21.x, k22.x);
        int gradient_g = sobel(k00.y, k01.y, k02.y, k10.y, k11.y, k12.y, k20.y, k21.y, k22.y);
        int gradient_b = sobel(k00.z, k01.z, k02.z, k10.z, k11.z, k12.z, k20.z, k21.z, k22.z);

        //int gradient = (gradient_r + gradient_g + gradient_b) / 3;

        pixel.x = gradient_r;
        pixel.y = gradient_g;
        pixel.z = gradient_b;
        pixel.w = 255;
    }
    output[__mul24(y, width) + x] = pixel;
}


int main(int argc, char** argv)
{
    unsigned char* h_data=NULL;
    unsigned char* d_input=NULL;
    uchar4* d_output=NULL;

    unsigned int height,width;

    int OUTPUTSIZE = atoi(argv[1]);
    unsigned int newheight=OUTPUTSIZE,newwidth=OUTPUTSIZE;

    //char file_name[]="./img/lena_rgba.ppm";
    char *file_name = argv[2];
    char* image_path = sdkFindFilePath(file_name, argv[0]);
    if(image_path==0)
        exit(0);

    printf("Open %s\n",image_path);
    sdkLoadPPM4(image_path, &h_data, &width, &height);
    int size = height*width*sizeof(unsigned char)*BPP;
    int newsize=newheight*newwidth*sizeof(uchar4);

    printf("Original Image Size: [%d, %d], size: %d\n", height, width, size);
    printf("New      Image Size: [%d, %d], size: %d\n", newheight, newwidth, newsize);

    checkCudaErrors(hipMalloc((void**)&d_input,size));
    checkCudaErrors(hipMemcpy(d_input, h_data, size, hipMemcpyHostToDevice));
    checkCudaErrors(hipMalloc((void**)&d_output,newsize));

    dim3 dimBlock(BLOCKSIZE, BLOCKSIZE, 1);
    dim3 dimGrid(newwidth / dimBlock.x, newheight / dimBlock.y, 1);

    StopWatchInterface *timer = NULL;
    sdkCreateTimer( &timer);
    sdkStartTimer( &timer);

    local_filter<<<dimGrid,dimBlock>>>(d_input,d_output,newwidth,newheight);

    checkCudaErrors( hipDeviceSynchronize() );
    sdkStopTimer( &timer);
    printf("Processing time: %f (ms)\n", sdkGetTimerValue( &timer));
    sdkDeleteTimer(&timer);

    unsigned char* h_odata;
    h_odata=(unsigned char*)malloc(newsize);
    checkCudaErrors(hipMemcpy(h_odata,d_output,newsize,hipMemcpyDeviceToHost));

    char outputpath[1024];
    strcpy(outputpath,image_path+6);
    strcpy(outputpath+strlen(image_path+6)-4,"_local_output.ppm");
    sdkSavePPM4ub( outputpath, h_odata, newwidth, newheight);
    printf("Wrote '%s'\n\n", outputpath);

    checkCudaErrors(hipFree(d_input));
    checkCudaErrors(hipFree(d_output));
    free(image_path);
    free(h_data);
    free(h_odata);
}
