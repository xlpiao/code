#include "hip/hip_runtime.h"
// System includes
#include <assert.h>
#include <stdio.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// helper functions and utilities to work with CUDA
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

#ifndef MAX
#define MAX(a, b) (a > b ? a : b)
#endif

__global__ void testKernel(int val) {
  cuPrintf("Hello, world from the device!\n");
  cuPrintf("[%d, %d]:\t\tValue is:%d\n", blockIdx.y * gridDim.x + blockIdx.x,
           threadIdx.z * blockDim.x * blockDim.y + threadIdx.y * blockDim.x +
               threadIdx.x,
           val);
}

int main(int argc, char **argv) {
  int devID;
  hipDeviceProp_t props;

  printf("\n*****Hello, world from the host!*****\n\n");

  // This will pick the best possible CUDA capable device
  devID = findCudaDevice(argc, (const char **)argv);

  // Get GPU information
  checkCudaErrors(hipGetDevice(&devID));
  checkCudaErrors(hipGetDeviceProperties(&props, devID));
  printf("Device %d: \"%s\" with Compute %d.%d capability\n", devID, props.name,
         props.major, props.minor);

  printf("printf() is called. Output:\n\n");

  // Kernel configuration, where a two-dimensional grid and
  // three-dimensional blocks are configured.
  dim3 dimGrid(2, 2);
  dim3 dimBlock(2, 2, 2);
  testKernel<<<dimGrid, dimBlock>>>(10);
  hipDeviceSynchronize();

  // hipDeviceReset causes the driver to clean up all state. While
  // not mandatory in normal operation, it is good practice.  It is also
  // needed to ensure correct operation when the application is being
  // profiled. Calling hipDeviceReset causes all profile data to be
  // flushed before the application exits
  hipDeviceReset();

  return EXIT_SUCCESS;
}
