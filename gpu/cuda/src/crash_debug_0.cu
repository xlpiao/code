#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

__global__ void foo(int *ptr)
{
  *ptr = 7;
}

int main(void)
{
  foo<<<1,1>>>(0);

  // make the host block until the device is finished with foo
  hipDeviceSynchronize();

  // check for error
  hipError_t error = hipGetLastError();
  if(error != hipSuccess)
  {
    // print the CUDA error message and exit
    printf("CUDA error: %s\n", hipGetErrorString(error));
    exit(-1);
  }

  return 0;
}
