
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void colonel(int *a_d){
  atomicAdd( a_d, blockIdx.x * blockDim.x + threadIdx.x);
}

int main(){

  int a = 0, *a_d;
  
  hipMalloc((void**) &a_d, sizeof(int));
  hipMemcpy(a_d, &a, sizeof(int), hipMemcpyHostToDevice);

  float   elapsedTime;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord( start, 0 );

  colonel<<<4,4>>>(a_d); // global id: 0 ~ 15, atomicAdd = sum(0+1+2+3+...+15) 
  
  hipEventRecord( stop, 0 );
  hipEventSynchronize( stop );
  hipEventElapsedTime( &elapsedTime, start, stop );
  hipEventDestroy( start );
  hipEventDestroy( stop );
  printf("GPU Time elapsed: %f seconds\n", elapsedTime/1000.0);
  
  
  hipMemcpy(&a, a_d, sizeof(int), hipMemcpyDeviceToHost);

  printf("a = %d\n", a);
  hipFree(a_d);

}
