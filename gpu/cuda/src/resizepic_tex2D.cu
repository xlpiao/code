#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include "hip/hip_runtime_api.h"
#include "helper_functions.h"

#ifndef _RESIZEPIC_KERNEL_H_
#define _RESIZEPIC_KERNEL_H_

texture<float,2,hipReadModeElementType> texRef;

__global__ void resizePic(float* output,int width, int height)
{
     int x= blockIdx.x * blockDim.x + threadIdx.x;
     int y= blockIdx.y * blockDim.y + threadIdx.y;

    float u = x/(float)width;
    float v = y/(float)height;

    // Read from texture and write to global memory
    output[y*width+x]=tex2D(texRef,u,v);
}

#endif


#define TARGET_SIZE 1024
char* file_name=(char*)"lena_bw.pgm";

int main(int argc, char** argv)
{
    float* h_data=NULL;
    float* d_data=NULL;

    unsigned int height,width;        //原始大小
    unsigned int newheight=TARGET_SIZE,newwidth=TARGET_SIZE;        //拉伸大小

    //开始读取图片，使用cuda的读PGM函数
    char* image_path = sdkFindFilePath(file_name, argv[0]);
    if(image_path==0)
        exit(0);

    printf("Open %s\n",image_path);
    sdkLoadPGM(image_path, &h_data, &width, &height);
    int size = height*width*sizeof(float);
    int newsize=newheight*newwidth*sizeof(float);

    printf("Original Image Size: [%-4d, %-4d], size: %d\n", height, width, size);
    printf("Target   Image Size: [%-4d, %-4d], size: %d\n", newheight, newwidth, newsize);


    checkCudaErrors(hipMalloc((void**)&d_data,newsize));

    //为CUDA数组分配内存，并将输入图像拷贝到内存
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32,0,0,0,hipChannelFormatKindFloat);
    hipArray* cuArray;
    checkCudaErrors(hipMallocArray(&cuArray,&channelDesc,width,height));
    checkCudaErrors(hipMemcpyToArray(cuArray,0,0,h_data,size,hipMemcpyHostToDevice));

    //设置纹理参数
    texRef.addressMode[0]=hipAddressModeWrap;
    texRef.addressMode[1]=hipAddressModeWrap;
    texRef.filterMode=hipFilterModeLinear;
    texRef.normalized=true;

    //纹理和数组绑定
    checkCudaErrors(hipBindTextureToArray(texRef,cuArray,channelDesc));

    //开始计算
    dim3 dimBlock(8, 8, 1);
    dim3 dimGrid(newwidth / dimBlock.x, newheight / dimBlock.y, 1);

    StopWatchInterface *timer = NULL;
    sdkCreateTimer( &timer);
    sdkStartTimer( &timer);

    resizePic<<<dimGrid,dimBlock>>>(d_data,newwidth,newheight);

    checkCudaErrors( hipDeviceSynchronize() );
    sdkStopTimer( &timer);
    printf("Processing time: %f (ms)\n", sdkGetTimerValue( &timer));
    sdkDeleteTimer(&timer);

    //拷贝结果，并存储
    float* h_odata;
    h_odata=(float*)malloc(newsize);
    checkCudaErrors(hipMemcpy(h_odata,d_data,newsize,hipMemcpyDeviceToHost));

    char outputpath[1024];
    strcpy(outputpath,image_path);
    strcpy(outputpath+strlen(image_path)-4,"_output.pgm");
    sdkSavePGM( outputpath, h_odata, newwidth, newheight);
    printf("Wrote '%s'\n", outputpath);

    checkCudaErrors(hipFree(d_data));
    checkCudaErrors(hipFreeArray(cuArray));
    free(image_path);
    free(h_data);
    free(h_odata);
}
