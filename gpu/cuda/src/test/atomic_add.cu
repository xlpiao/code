
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void colonel(int *d_a) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  printf("Before %d, %d, %d, index = %d, *d_a = %d\n", blockIdx.x, blockDim.x,
         threadIdx.x, index, *d_a);
  atomicAdd(d_a, index);
  printf("After %d, %d, %d, index = %d, *d_a = %d\n", blockIdx.x, blockDim.x,
         threadIdx.x, index, *d_a);
}

int main() {
  int h_a = 0, *d_a;

  hipMalloc((void **)&d_a, sizeof(int));
  hipMemcpy(d_a, &h_a, sizeof(int), hipMemcpyHostToDevice);

  float elapsedTime;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  //// 1D
  colonel<<<4, 4>>>(d_a);  // global id: 0 ~ 15, atomicAdd = sum(0+1+2+3+...+15)

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start, stop);
  hipEventDestroy(start);
  hipEventDestroy(stop);
  printf("GPU Time elapsed: %f seconds\n", elapsedTime / 1000.0);

  hipMemcpy(&h_a, d_a, sizeof(int), hipMemcpyDeviceToHost);

  printf("h_a = %d\n", h_a);
  hipFree(d_a);
}
