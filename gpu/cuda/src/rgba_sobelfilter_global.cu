#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include <stdio.h>
#include <string.h>

#define BPP 4

#define BLOCKSIZE 16

__shared__ short int Gx[3][3];
__shared__ short int Gy[3][3];

__device__ int sobel(unsigned char k00, unsigned char k01, unsigned char k02,
                     unsigned char k10, unsigned char k11, unsigned char k12,
                     unsigned char k20, unsigned char k21, unsigned char k22) {
  Gx[0][0] = -1;
  Gx[0][1] = 0;
  Gx[0][2] = 1;
  Gx[1][0] = -2;
  Gx[1][1] = 0;
  Gx[1][2] = 2;
  Gx[2][0] = -1;
  Gx[2][1] = 0;
  Gx[2][2] = 1;
  Gy[0][0] = 1;
  Gy[0][1] = 2;
  Gy[0][2] = 1;
  Gy[1][0] = 0;
  Gy[1][1] = 0;
  Gy[1][2] = 0;
  Gy[2][0] = -1;
  Gy[2][1] = -2;
  Gy[2][2] = -1;
  int pixelX = k00 * Gx[0][0] + k01 * Gx[0][1] + k02 * Gx[0][2] +
               k10 * Gx[1][0] + k11 * Gx[1][1] + k12 * Gx[1][2] +
               k20 * Gx[2][0] + k21 * Gx[2][1] + k22 * Gx[2][2];
  int pixelY = k00 * Gy[0][0] + k01 * Gy[0][1] + k02 * Gy[0][2] +
               k10 * Gy[1][0] + k11 * Gy[1][1] + k12 * Gy[1][2] +
               k20 * Gy[2][0] + k21 * Gy[2][1] + k22 * Gy[2][2];
  int pixel = abs(pixelX) + abs(pixelY);
  pixel = (pixel < 0) ? 0 : pixel;
  pixel = (pixel > 255) ? 255 : pixel;

  return pixel;
}

__device__ int getindexForPixelAt(int x, int y, int width, int height) {
  int val = BPP * (x + y * width);
  const int max = width * height * BPP - 1;
  if (val < 0) {
    return 0;
  }
  if (val > max) {
    return max;
  }
  return val;
}

__device__ uchar4 pixelAt(unsigned char* input, int x, int y, int width,
                          int height) {
  int start = getindexForPixelAt(x, y, width, height);
  uchar4 pixel = {input[start], input[start + 1], input[start + 2],
                  input[start + 3]};
  ;

  return pixel;
}

__global__ void global_filter(unsigned char* input, uchar4* output, int width,
                              int height) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  uchar4 k00, k01, k02, k10, k11, k12, k20, k21, k22;
  uchar4 pixel = {0, 0, 0, 255};
  // pixel.x = 0, pixel.y = 0, pixel.z = 0, pixel.w = 255;

  if (x - 1 >= 0 && x + 1 < height && y - 1 >= 0 && y + 1 < width) {
    k00 = pixelAt(input, x - 1, y - 1, width, height);
    k01 = pixelAt(input, x - 1, y, width, height);
    k02 = pixelAt(input, x - 1, y + 1, width, height);

    k10 = pixelAt(input, x, y - 1, width, height);
    k11 = pixelAt(input, x, y, width, height);
    k12 = pixelAt(input, x, y + 1, width, height);

    k20 = pixelAt(input, x + 1, y - 1, width, height);
    k21 = pixelAt(input, x + 1, y, width, height);
    k22 = pixelAt(input, x + 1, y + 1, width, height);

    int gradient_r =
        sobel(k00.x, k01.x, k02.x, k10.x, k11.x, k12.x, k20.x, k21.x, k22.x);
    int gradient_g =
        sobel(k00.y, k01.y, k02.y, k10.y, k11.y, k12.y, k20.y, k21.y, k22.y);
    int gradient_b =
        sobel(k00.z, k01.z, k02.z, k10.z, k11.z, k12.z, k20.z, k21.z, k22.z);

    // int gradient = (gradient_r + gradient_g + gradient_b) / 3;

    pixel.x = gradient_r;
    pixel.y = gradient_g;
    pixel.z = gradient_b;
    pixel.w = 255;
  }
  output[__mul24(y, width) + x] = pixel;
}

int main(int argc, char** argv) {
  unsigned char* h_data = NULL;
  unsigned char* d_input = NULL;
  uchar4* d_output = NULL;

  unsigned int height, width;

  int OUTPUTSIZE = atoi(argv[1]);
  unsigned int newheight = OUTPUTSIZE, newwidth = OUTPUTSIZE;

  // char file_name[]="./img/lena_rgba.ppm";
  char* file_name = argv[2];
  char* image_path = sdkFindFilePath(file_name, argv[0]);
  if (image_path == 0) exit(0);

  printf("Open %s\n", image_path);
  sdkLoadPPM4(image_path, &h_data, &width, &height);
  int size = height * width * sizeof(unsigned char) * BPP;
  int newsize = newheight * newwidth * sizeof(uchar4);

  printf("Original Image Size: [%d, %d], size: %d\n", height, width, size);
  printf("New      Image Size: [%d, %d], size: %d\n", newheight, newwidth,
         newsize);

  checkCudaErrors(hipMalloc((void**)&d_input, size));
  checkCudaErrors(hipMemcpy(d_input, h_data, size, hipMemcpyHostToDevice));
  checkCudaErrors(hipMalloc((void**)&d_output, newsize));

  dim3 dimBlock(BLOCKSIZE, BLOCKSIZE, 1);
  dim3 dimGrid(newwidth / dimBlock.x, newheight / dimBlock.y, 1);

  StopWatchInterface* timer = NULL;
  sdkCreateTimer(&timer);
  sdkStartTimer(&timer);

  global_filter<<<dimGrid, dimBlock>>>(d_input, d_output, newwidth, newheight);

  checkCudaErrors(hipDeviceSynchronize());
  sdkStopTimer(&timer);
  printf("Processing time: %f (ms)\n", sdkGetTimerValue(&timer));
  sdkDeleteTimer(&timer);

  unsigned char* h_odata;
  h_odata = (unsigned char*)malloc(newsize);
  checkCudaErrors(
      hipMemcpy(h_odata, d_output, newsize, hipMemcpyDeviceToHost));

  char outputpath[1024];
  strcpy(outputpath, image_path + 6);
  strcpy(outputpath + strlen(image_path + 6) - 4, "_global_output.ppm");
  sdkSavePPM4ub(outputpath, h_odata, newwidth, newheight);
  printf("Wrote '%s'\n\n", outputpath);

  checkCudaErrors(hipFree(d_input));
  checkCudaErrors(hipFree(d_output));
  free(image_path);
  free(h_data);
  free(h_odata);
}
