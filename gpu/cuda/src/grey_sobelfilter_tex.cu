#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include <stdio.h>
#include <string.h>

#define BPP 1

#define BLOCKSIZE 16
texture<unsigned char, 2> tex;

__shared__ short int Gx[3][3];
__shared__ short int Gy[3][3];

__global__ void tex_filter(unsigned char* output, int width, int height) {
  Gx[0][0] = -1;
  Gx[0][1] = 0;
  Gx[0][2] = 1;
  Gx[1][0] = -2;
  Gx[1][1] = 0;
  Gx[1][2] = 2;
  Gx[2][0] = -1;
  Gx[2][1] = 0;
  Gx[2][2] = 1;
  Gy[0][0] = 1;
  Gy[0][1] = 2;
  Gy[0][2] = 1;
  Gy[1][0] = 0;
  Gy[1][1] = 0;
  Gy[1][2] = 0;
  Gy[2][0] = -1;
  Gy[2][1] = -2;
  Gy[2][2] = -1;
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  int pixel = 0, pixelX = 0, pixelY = 0;

  if (x - 1 >= 0 && x + 1 < height && y - 1 >= 0 && y + 1 < width) {
    for (int i = -1; i <= 1; i++) {
      for (int j = -1; j <= 1; j++) {
        pixelX += (int)(tex2D(tex, x + i, y + j) * Gx[i + 1][j + 1]);
        pixelY += (int)(tex2D(tex, x + i, y + j) * Gy[i + 1][j + 1]);
      }
    }
  }

  pixel = abs(pixelX) + abs(pixelY);
  pixel = (pixel < 0) ? 0 : pixel;
  pixel = (pixel > 255) ? 255 : pixel;
  output[x + y * width] = pixel;

  // Read from texture and write to global memory
  // output[y*width+x]=input[y*width+x];
}

int main(int argc, char** argv) {
  unsigned char* h_data = NULL;
  hipArray* d_input = NULL;
  unsigned char* d_output = NULL;

  unsigned int height, width;

  int OUTPUTSIZE = atoi(argv[1]);
  unsigned int newheight = OUTPUTSIZE, newwidth = OUTPUTSIZE;

  // char file_name[]="./img/lena_grey.pgm";
  char* file_name = argv[2];
  char* image_path = sdkFindFilePath(file_name, argv[0]);
  if (image_path == 0) exit(0);

  printf("Open %s\n", image_path);
  sdkLoadPGM(image_path, &h_data, &width, &height);
  int size = height * width * sizeof(unsigned char) * BPP;
  int newsize = newheight * newwidth * sizeof(unsigned char) * BPP;

  printf("Original Image Size: [%d, %d], size: %d\n", height, width, size);
  printf("Target   Image Size: [%d, %d], size: %d\n", newheight, newwidth,
         newsize);

  checkCudaErrors(hipMalloc((void**)&d_output, newsize));

  hipChannelFormatDesc channelDesc = hipCreateChannelDesc<unsigned char>();
  checkCudaErrors(hipMallocArray(&d_input, &channelDesc, width, height));
  checkCudaErrors(
      hipMemcpyToArray(d_input, 0, 0, h_data, size, hipMemcpyHostToDevice));

  checkCudaErrors(hipBindTextureToArray(tex, d_input, channelDesc));

  dim3 dimBlock(BLOCKSIZE, BLOCKSIZE, 1);
  dim3 dimGrid(newwidth / dimBlock.x, newheight / dimBlock.y, 1);

  StopWatchInterface* timer = NULL;
  sdkCreateTimer(&timer);
  sdkStartTimer(&timer);

  tex_filter<<<dimGrid, dimBlock>>>(d_output, newwidth, newheight);

  checkCudaErrors(hipDeviceSynchronize());
  sdkStopTimer(&timer);
  printf("Processing time: %f (ms)\n", sdkGetTimerValue(&timer));
  sdkDeleteTimer(&timer);

  unsigned char* h_odata;
  h_odata = (unsigned char*)malloc(newsize);
  checkCudaErrors(
      hipMemcpy(h_odata, d_output, newsize, hipMemcpyDeviceToHost));

  char outputpath[1024];
  strcpy(outputpath, image_path + 6);
  strcpy(outputpath + strlen(image_path + 6) - 4, "_tex_output.pgm");
  sdkSavePGM(outputpath, h_odata, newwidth, newheight);
  printf("Wrote '%s'\n\n", outputpath);

  checkCudaErrors(hipFreeArray(d_input));
  checkCudaErrors(hipFree(d_output));
  free(image_path);
  free(h_data);
  free(h_odata);
}
