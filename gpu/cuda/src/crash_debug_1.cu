
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

int main(void) {
  int *ptr = 0;

  // gimme!
  hipError_t error = hipMalloc((void **)&ptr, UINT_MAX);
  if (error != hipSuccess) {
    // print the CUDA error message and exit
    printf("CUDA error: %s\n", hipGetErrorString(error));
    exit(-1);
  }

  return 0;
}
